
#include <hip/hip_runtime.h>
#include <iostream>
#define N 100

__global__ void CUDAAdd1(int a, int b, int *c) {
    *c = a + b;
}

void CPUAdd(int *a, int *b, int *c) {
    int tid = 0;
    while (tid < N) {
        c[tid] = a[tid] + b[tid];
        tid++;
    }
}

void test01() {
    // 申请内存
    int c;      // Host端数据内存
    int *dev_c; // device端数据内存
    hipMalloc((void**)&dev_c, sizeof(int));

    // 执行核函数计算
    CUDAAdd1<<<1,1>>>(2, 7, dev_c);// 注意尖括号个数

    // 拷贝计算结果,释放GPU内存
    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_c);

    std::cout << "2 + 7 = " << c << std::endl;
}

void test02()
{
    int a[N], b[N], c[N];

    // 赋值
    for (int i = 0; i < N; i++) {
        a[i] = -i + i * i;
        b[i] = i * i * i;
    }

    // CPU求和
    CPUAdd(a, b, c);

    for (int i = 0; i < N; i++)
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
}

int main(int argc, char const *argv[]) {
    test01();
    test02();
    return 0;
}
